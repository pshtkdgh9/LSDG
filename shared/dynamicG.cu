#include "subgraph.cuh"
#include "gpu_error_check.cuh"
#include "graph.cuh"
#include <hip/hip_runtime_api.h>

template <class E>
dynamicGraph<E>::dynamicGraph(string graphFilePath, bool isWeighted)
{
	this->dynamicgraphFilePath = graphFilePath;
    this->dynamicisWeighted = isWeighted;
    
    if(graphFormat == "bcsr" || graphFormat == "bwcsr")
	{
		ifstream infile (dynamicgraphFilePath, ios::in | ios::binary);
	
		infile.read ((char*)&num_nodes, sizeof(uint));
		infile.read ((char*)&num_edges, sizeof(uint));
		
		nodePointer = new uint[num_nodes+1];
		gpuErrorcheck(hipHostMalloc(&edgeList, (num_edges) * sizeof(E)));   //edgelist作为cudamallochost  锁页内存？
		
		infile.read ((char*)nodePointer, sizeof(uint)*num_nodes);
		infile.read ((char*)edgeList, sizeof(E)*num_edges);
		nodePointer[num_nodes] = num_edges;
    }
    ///add
}