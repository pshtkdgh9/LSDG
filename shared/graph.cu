#include "hip/hip_runtime.h"
#include "graph.cuh"
#include "gpu_error_check.cuh"

template <class E>
Graph<E>::Graph(string graphFilePath, bool isWeighted)
{
	this->graphFilePath = graphFilePath;
	this->isWeighted = isWeighted;
}

template <class E>
string Graph<E>::GetFileExtension(string fileName)
{
    if(fileName.find_last_of(".") != string::npos)
        return fileName.substr(fileName.find_last_of(".")+1);
    return "";
}


template <>
void Graph<OutEdgeWeighted>::AssignW8(uint w8, uint index)
{
	int sourceNode; //수정
    int destinationNode; //수정
    edgeList[index].w8 = w8;
}

template <>
void Graph<OutEdge>::AssignW8(uint w8, uint index)
{
    edgeList[index].end = edgeList[index].end; // do nothing
}

template <class E>
void Graph<E>::ReadGraph()
{

	cout << "Reading the input graph from the following file:\n>> " << graphFilePath << endl;
	
	this->graphFormat = GetFileExtension(graphFilePath);
	
	if(graphFormat == "el" || graphFormat == "wel")
	{
		ifstream infile;
		infile.open(graphFilePath);
		stringstream ss;
		uint max = 0;
		string line;
		uint edgeCounter = 0;
		
		vector<Edge> edges;
		Edge newEdge;
		while(getline( infile, line ))
		{
			ss.str("");
			ss.clear();
			ss << line;
			
			ss >> newEdge.source;
			ss >> newEdge.end;
			
			edges.push_back(newEdge);
			edgeCounter++;
			
			if(max < newEdge.source)
				max = newEdge.source;
			if(max < newEdge.end)
				max = newEdge.end;				
		}
		infile.close();
		num_nodes = max + 1;
		num_edges = edgeCounter;
		nodePointer = new uint[num_nodes+1];
		gpuErrorcheck(hipHostMalloc(&edgeList, (num_edges) * sizeof(E)));// 边文件分配锁页内存
		uint *degree = new uint[num_nodes];
		for(uint i=0; i<num_nodes; i++)
			degree[i] = 0;
		for(uint i=0; i<num_edges; i++)
			degree[edges[i].source]++;
		
		uint counter=0;
		for(uint i=0; i<num_nodes; i++)
		{
			nodePointer[i] = counter;
			counter = counter + degree[i];
		}
		nodePointer[num_nodes] = num_edges;
		uint *outDegreeCounter  = new uint[num_nodes];
		uint location;  
		for(uint i=0; i<num_edges; i++)
		{
			location = nodePointer[edges[i].source] + outDegreeCounter[edges[i].source];
			edgeList[location].end = edges[i].end;
			//if(isWeighted)
			//	edgeList[location].w8 = edges[i].w8;
			outDegreeCounter[edges[i].source]++;  
		}
		edges.clear();
		delete[] degree;
		delete[] outDegreeCounter;						
		
	}
	else
	{
		cout << "The graph format is not supported!\n";
		exit(-1);
	}
	
	//outDegree  = new unsigned int[num_nodes];   //分配内存
	hipHostAlloc((void**)&outDegree,num_nodes*sizeof(unsigned int),hipHostMallocDefault);    //newadd
	printf("2222222\n");
	for(uint i=1; i<num_nodes-1; i++)
		outDegree[i-1] = nodePointer[i] - nodePointer[i-1];
	outDegree[num_nodes-1] = num_edges - nodePointer[num_nodes-1];
	
	 //label1 = new bool[num_nodes];
	 //label2 = new bool[num_nodes];
	hipHostAlloc((void**)&label1,num_nodes*sizeof(bool),hipHostMallocDefault);  //newadd
	hipHostAlloc((void**)&label2,num_nodes*sizeof(bool),hipHostMallocDefault);  // newadd
	//value  = new unsigned int[num_nodes];
	hipHostAlloc((void**)&value,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value1,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value2,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value3,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value4,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value5,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value6,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd
	hipHostAlloc((void**)&value7,num_nodes*sizeof(unsigned int),hipHostMallocDefault); //newadd

	gpuErrorcheck(hipMalloc(&d_outDegree, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value1, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value2, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value3, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value4, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value5, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value6, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_value7, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_label1, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label11, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label22, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label111, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label222, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label1111, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label2222, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label5, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label6, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label55, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label66, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label555, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label666, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label5555, num_nodes * sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_label6666, num_nodes * sizeof(bool)));
	
	cout << "Done reading.\n";
	cout << "Number of nodes = " << num_nodes << endl;
	cout << "Number of edges = " << num_edges << endl;
	
	/*
	for(int i=0; i<10; i++)
		cout << nodePointer[i] << " ";
	cout << "\n\n\n\n\n";
	
	cout <<  nodePointer[1] - nodePointer[0] << endl;
	for(int i=nodePointer[0]; i<nodePointer[1]; i++)
		cout << edgeList[i].end << " ";
	cout << "\n\n\n\n\n";
	
	cout <<  nodePointer[100001] - nodePointer[100000] << endl;
	for(int i=nodePointer[100000]; i<nodePointer[100001]; i++)
		cout << edgeList[i].end << " ";
	cout << "\n\n\n\n\n";
	
	cout <<  nodePointer[1000001] - nodePointer[1000000] << endl;
	for(int i=nodePointer[1000000]; i<nodePointer[1000001]; i++)
		cout << edgeList[i].end << " ";	
	cout << "\n\n\n\n\n";
	
	cout <<  nodePointer[num_nodes] - nodePointer[num_nodes-1] << endl;
	for(int i=nodePointer[num_nodes-1]; i<nodePointer[num_nodes]; i++)
		cout << edgeList[i].end << " ";
	cout << "\n\n\n\n\n";
	*/

}


template class Graph<OutEdge>;
template class Graph<OutEdgeWeighted>;
