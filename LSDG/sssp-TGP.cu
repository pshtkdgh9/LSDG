#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/egraph_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"


int main(int argc, char** argv)
{
	/*
	Test<int> test;
	cout << test.sum(20, 30) << endl;
	*/
	hipFree(0);
	ArgumentParser arguments(argc, argv, true, false);//路径，是否weight
	
	Timer timer,timer1,timer2;   //Timer类
	timer.Start();
	timer1.Start();
	
	Graph<OutEdgeWeighted> graph(arguments.input, true);
	graph.ReadGraph();
	//graph.dynamicReadGraph();
	float readtime = timer.Finish();
	cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	//for(unsigned int i=0; i<100; i++)
	//	cout << graph.edgeList[i].end << " " << graph.edgeList[i].w8;

			//multi stream
     		hipDeviceProp_t  prop;
			int whichDevice;
			hipGetDevice( &whichDevice );
		    hipGetDeviceProperties( &prop, whichDevice );
			if (!prop.deviceOverlap) {
				printf( "Device will not handle overlaps, so no speed up from streams\n" );
				return 0;
			}

			hipEvent_t start, stop;
			float elapsedTime;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start, 0);

			hipStream_t stream0,stream1,stream2,stream3,stream4,stream5,stream6,stream7;
			hipStreamCreate(&stream0);
			hipStreamCreate(&stream1); 
			hipStreamCreate(&stream2);
			hipStreamCreate(&stream3); 
			hipStreamCreate(&stream4);
			hipStreamCreate(&stream5); 
			hipStreamCreate(&stream6);
			hipStreamCreate(&stream7); 
			//end

			
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.value[i] = DIST_INFINITY;
		graph.value1[i]=DIST_INFINITY;
		graph.value2[i] = DIST_INFINITY;
		graph.value3[i]=DIST_INFINITY;
		graph.value4[i] = DIST_INFINITY;
		graph.value5[i]=DIST_INFINITY;
		graph.value6[i] = DIST_INFINITY;
		graph.value7[i]=DIST_INFINITY;
		graph.label1[i] = true;
		graph.label2[i] = false;
	}
	graph.value[arguments.sourceNode] = 0;
	//graph.label[arguments.sourceNode] = true;

	 //gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice)); //these four GPU design on graph.cu with cudamalloc
	 //gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
	 //gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	 //gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));

	hipMemcpyAsync(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream0);// newadd
	hipMemcpyAsync(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream0);// newadd
	hipMemcpyAsync(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream0);// newadd
	hipMemcpyAsync(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream0);// newadd

	//hipMemcpyAsync(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream1);// newadd
	hipMemcpyAsync(graph.d_value1, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream1);// newadd
	hipMemcpyAsync(graph.d_label11, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream1);// newadd
	hipMemcpyAsync(graph.d_label22, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream1);// newadd

	hipMemcpyAsync(graph.d_value2, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream2);// newadd
	hipMemcpyAsync(graph.d_label111, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream2);// newadd
	hipMemcpyAsync(graph.d_label222, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream2);// newadd

	hipMemcpyAsync(graph.d_value3, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream3);// newadd
	hipMemcpyAsync(graph.d_label1111, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream3);// newadd
	hipMemcpyAsync(graph.d_label2222, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream3);// newadd

	hipMemcpyAsync(graph.d_value4, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream4);// newadd
	hipMemcpyAsync(graph.d_label5, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream4);// newadd
	hipMemcpyAsync(graph.d_label6, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream4);// newadd

	hipMemcpyAsync(graph.d_value5, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream5);// newadd
	hipMemcpyAsync(graph.d_label55, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream5);// newadd
	hipMemcpyAsync(graph.d_label66, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream5);// newadd

	hipMemcpyAsync(graph.d_value6, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream6);// newadd
	hipMemcpyAsync(graph.d_label555, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream6);// newadd
	hipMemcpyAsync(graph.d_label666, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream6);// newadd

	hipMemcpyAsync(graph.d_value7, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice,stream7);// newadd
	hipMemcpyAsync(graph.d_label5555, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream7);// newadd
	hipMemcpyAsync(graph.d_label6666, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice,stream7);// newadd


	//printf("1111\n");
	Subgraph<OutEdgeWeighted> subgraph(graph.num_nodes, graph.num_edges);//duixiang
	
	SubgraphGenerator<OutEdgeWeighted> subgen(graph); //duixinag
	//printf("*****************\n");
	subgen.generate(graph, subgraph);//对象.generate(图，子图)  -------> 生成graph的子图
	//printf("*****************\n");
	for(unsigned int i=0; i<graph.num_nodes; i++)
	{
		graph.label1[i] = false;
	}
	graph.label1[arguments.sourceNode] = true;
	//gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));	

	gpuErrorcheck(hipMemcpyAsync(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream0));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label11, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream1));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label111, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream2));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label1111, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream3));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label5, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream4));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label55, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream5));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label555, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream6));	
	gpuErrorcheck(hipMemcpyAsync(graph.d_label5555, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice, stream7));

	hipDeviceSynchronize();
	Partitioner<OutEdgeWeighted> partitioner;
	//printf("*****************##############\n");

	timer.Start();
	
	uint gItr = 0;
	
	bool finished,finished1,finished2,finished3,finished4,finished5,finished6,finished7;
	bool *d_finished;
	bool *d_finished1;
	bool *d_finished2;
	bool *d_finished3;
	bool *d_finished4;
	bool *d_finished5;
	bool *d_finished6;
	bool *d_finished7;
	clock_t start1;
	gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished1, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished2, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished3, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished4, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished5, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished6, sizeof(bool)));
	gpuErrorcheck(hipMalloc(&d_finished7, sizeof(bool)));
	//hipHostMalloc((void**)&finished,sizeof(bool));
	while (subgraph.numActiveNodes>0)    //整个子图
	{
		gItr++;//全局迭代次数
		//printf("test******************\n");
		partitioner.partition(subgraph, subgraph.numActiveNodes);// Subgraph分区
		// for(int i=0; i<partitioner.numPartitions; i++)
		// {
		// 	if(i==2||i==4||i==8)
		// 	partitioner.dynamicGraph(partitioner.partitionNodeSize[i],partitioner.fromNode[i],partitioner.fromEdge[i]);//dynamic分区的构造
		// 	//partitioner.dynamicpartition()//传第二分区
		// }
		cout<<"numPartitions=="<<partitioner.numPartitions<<endl;
		// a super iteration
		
		for(int i=0; i<partitioner.numPartitions; i++) //<
		{
			if(i==1||i==2||i==3||i==4||i==5)
			{
				hipDeviceSynchronize();
				gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream1));
				hipDeviceSynchronize();
				mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream1>>>(subgraph.d_activeNodes, graph.d_label11, graph.d_label22, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
				hipDeviceSynchronize();
				uint itr = 0;
				do
			{
				itr++;
				finished1=true;
				gpuErrorcheck(hipMemcpyAsync(d_finished1, &finished1, sizeof(bool), hipMemcpyHostToDevice,stream1));
				//gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
				hipDeviceSynchronize();

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1,256,0,stream1>>>(partitioner.partitionNodeSize[i],//host   partitioner.partitionNodeSize[i]/512 + 1 , 512
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value1, //cudamemcpy
						d_finished1,//wait
						(itr%2==1) ? graph.d_label11 : graph.d_label22,//cudamemcpy
						(itr%2==1) ? graph.d_label22 : graph.d_label11);		//cudamemcpy
				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				gpuErrorcheck(hipMemcpy(&finished1, d_finished1, sizeof(bool), hipMemcpyDeviceToHost));
				hipDeviceSynchronize();
				hipStreamSynchronize(stream1);
					//printf("^^^^^^^^^^^^^^^^^^^^\n");
					//cout<<"finished/finished1="<<finished<<""<<finished1<<endl;
			}while(!(finished1)); // finished=ture，finished=false才会循环 ,  ||finished1
			hipDeviceSynchronize();
			hipStreamSynchronize(stream1);
			// hipStreamSynchronize(stream1);
			cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
			}

				if(i==2)
				{
					hipDeviceSynchronize();
					gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream2));
					hipDeviceSynchronize();
					mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream2>>>(subgraph.d_activeNodes, graph.d_label111, graph.d_label222, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
					hipDeviceSynchronize();
					uint itr = 0;
					do
				{
					itr++;
					finished2 = true;
					gpuErrorcheck(hipMemcpyAsync(d_finished2, &finished2, sizeof(bool), hipMemcpyHostToDevice,stream2));
					//gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
					hipDeviceSynchronize();
	
						sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1,256,0,stream2>>>(partitioner.partitionNodeSize[i],//host   partitioner.partitionNodeSize[i]/512 + 1 , 512
							partitioner.fromNode[i],//host
							partitioner.fromEdge[i],//host
							subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
							subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
							subgraph.d_activeEdgeList,  //cudamemcpy
							graph.d_outDegree,//cudamemcpy
							graph.d_value2, //cudamemcpy
							d_finished2,//wait
							(itr%2==1) ? graph.d_label111 : graph.d_label222,//cudamemcpy
							(itr%2==1) ? graph.d_label222 : graph.d_label111);		//cudamemcpy
					hipDeviceSynchronize();
					gpuErrorcheck( hipPeekAtLastError() );	
					gpuErrorcheck(hipMemcpy(&finished2, d_finished2, sizeof(bool), hipMemcpyDeviceToHost));
					hipDeviceSynchronize();
					hipStreamSynchronize(stream2);
						//printf("^^^^^^^^^^^^^^^^^^^^\n");
						//cout<<"finished/finished1="<<finished<<""<<finished1<<endl;
				}while(!(finished2)); // finished=ture，finished=false才会循环 ,  ||finished1
				hipDeviceSynchronize();
				hipStreamSynchronize(stream2);
				// hipStreamSynchronize(stream1);
				cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
				}	


			if(i==3)
			{
				hipDeviceSynchronize();
				gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream1));
				hipDeviceSynchronize();
				mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream3>>>(subgraph.d_activeNodes, graph.d_label1111, graph.d_label2222, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
				hipDeviceSynchronize();
				uint itr = 0;
				do
			{
				itr++;
				finished1=true;
				gpuErrorcheck(hipMemcpyAsync(d_finished3, &finished3, sizeof(bool), hipMemcpyHostToDevice,stream1));
				//gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
				hipDeviceSynchronize();

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1,256,0,stream1>>>(partitioner.partitionNodeSize[i],//host   partitioner.partitionNodeSize[i]/512 + 1 , 512
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value3, //cudamemcpy
						d_finished3,//wait
						(itr%2==1) ? graph.d_label1111 : graph.d_label2222,//cudamemcpy
						(itr%2==1) ? graph.d_label2222 : graph.d_label1111);		//cudamemcpy
				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				gpuErrorcheck(hipMemcpy(&finished3, d_finished3, sizeof(bool), hipMemcpyDeviceToHost));
				hipDeviceSynchronize();
				hipStreamSynchronize(stream3);
					//printf("^^^^^^^^^^^^^^^^^^^^\n");
					//cout<<"finished/finished1="<<finished<<""<<finished1<<endl;
			}while(!(finished3)); // finished=ture，finished=false才会循环 ,  ||finished1
			hipDeviceSynchronize();
			hipStreamSynchronize(stream3);
			// hipStreamSynchronize(stream1);
			cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
			}



		if(i==0)
		{			
			gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream0));
			//[gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream1));
			//******** CPU-GPU的data transfer
			//hipDeviceSynchronize();
			//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label1,graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream0>>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream1>>>(subgraph.d_activeNodes, graph.d_label11, graph.d_label22, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 , 256,0,stream2>>>(subgraph.d_activeNodes, graph.d_label111, graph.d_label222, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 , 256,0,stream3>>>(subgraph.d_activeNodes, graph.d_label1111, graph.d_label2222, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream4>>>(subgraph.d_activeNodes, graph.d_label5, graph.d_label6, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 ,256,0,stream5>>>(subgraph.d_activeNodes, graph.d_label55, graph.d_label66, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 , 256,0,stream6>>>(subgraph.d_activeNodes, graph.d_label555, graph.d_label666, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			mixLabels<<<partitioner.partitionNodeSize[i]/256 + 1 , 256,0,stream7>>>(subgraph.d_activeNodes, graph.d_label5555, graph.d_label6666, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
			hipDeviceSynchronize();
			uint itr = 0;
			do
			{
				itr++;
				finished = true,finished1=true,finished2=true,finished3=true,finished4= true,finished5=true,finished6=true,finished7=true;;
				gpuErrorcheck(hipMemcpyAsync(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice,stream0));
				gpuErrorcheck(hipMemcpyAsync(d_finished1, &finished1, sizeof(bool), hipMemcpyHostToDevice,stream1));
				gpuErrorcheck(hipMemcpyAsync(d_finished2, &finished2, sizeof(bool), hipMemcpyHostToDevice,stream2));
				gpuErrorcheck(hipMemcpyAsync(d_finished3, &finished3, sizeof(bool), hipMemcpyHostToDevice,stream3));
				gpuErrorcheck(hipMemcpyAsync(d_finished4, &finished4, sizeof(bool), hipMemcpyHostToDevice,stream4));
				gpuErrorcheck(hipMemcpyAsync(d_finished5, &finished5, sizeof(bool), hipMemcpyHostToDevice,stream5));
				gpuErrorcheck(hipMemcpyAsync(d_finished6, &finished6, sizeof(bool), hipMemcpyHostToDevice,stream6));
				gpuErrorcheck(hipMemcpyAsync(d_finished7, &finished7, sizeof(bool), hipMemcpyHostToDevice,stream7));
				//gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
				hipDeviceSynchronize();

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1,256,0,stream0>>>(partitioner.partitionNodeSize[i],//host   partitioner.partitionNodeSize[i]/512 + 1 , 512
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value, //cudamemcpy
						d_finished,//wait
						(itr%2==1) ? graph.d_label1 : graph.d_label2,//cudamemcpy
						(itr%2==1) ? graph.d_label2 : graph.d_label1);		//cudamemcpy
						
					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream1>>>(partitioner.partitionNodeSize[i],//host
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value1, //cudamemcpy
						d_finished1,//wait
						(itr%2==1) ? graph.d_label11 : graph.d_label22,//cudamemcpy
						(itr%2==1) ? graph.d_label22 : graph.d_label11);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream2>>>(partitioner.partitionNodeSize[i],//host
							partitioner.fromNode[i],//host
							partitioner.fromEdge[i],//host
							subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
							subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
							subgraph.d_activeEdgeList,  //cudamemcpy
							graph.d_outDegree,//cudamemcpy
							graph.d_value2, //cudamemcpy
							d_finished2,//wait
							(itr%2==1) ? graph.d_label111 : graph.d_label222,//cudamemcpy
							(itr%2==1) ? graph.d_label222 : graph.d_label111);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream3>>>(partitioner.partitionNodeSize[i],//host
								partitioner.fromNode[i],//host
								partitioner.fromEdge[i],//host
								subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
								subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
								subgraph.d_activeEdgeList,  //cudamemcpy
								graph.d_outDegree,//cudamemcpy
								graph.d_value3, //cudamemcpy
								d_finished3,//wait
								(itr%2==1) ? graph.d_label1111 : graph.d_label2222,//cudamemcpy
								(itr%2==1) ? graph.d_label2222 : graph.d_label1111);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream4>>>(partitioner.partitionNodeSize[i],//host
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value4, //cudamemcpy
						d_finished4,//wait
						(itr%2==1) ? graph.d_label5 : graph.d_label6,//cudamemcpy
						(itr%2==1) ? graph.d_label6 : graph.d_label5);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream5>>>(partitioner.partitionNodeSize[i],//host
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value5, //cudamemcpy
						d_finished5,//wait
						(itr%2==1) ? graph.d_label55 : graph.d_label66,//cudamemcpy
						(itr%2==1) ? graph.d_label66 : graph.d_label55);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream6>>>(partitioner.partitionNodeSize[i],//host
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value6, //cudamemcpy
						d_finished6,//wait
						(itr%2==1) ? graph.d_label555 : graph.d_label666,//cudamemcpy
						(itr%2==1) ? graph.d_label666 : graph.d_label555);//cudamemcpy

					sssp_async<<<partitioner.partitionNodeSize[i]/256 + 1, 256,0,stream7>>>(partitioner.partitionNodeSize[i],//host
						partitioner.fromNode[i],//host
						partitioner.fromEdge[i],//host
						subgraph.d_activeNodes,  //  不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeNodesPointer, //不需要memcpy赋值，只cudamalloc,通过核函数在GPU中赋值
						subgraph.d_activeEdgeList,  //cudamemcpy
						graph.d_outDegree,//cudamemcpy
						graph.d_value7, //cudamemcpy
						d_finished7,//wait
						(itr%2==1) ? graph.d_label5555 : graph.d_label6666,//cudamemcpy
						(itr%2==1) ? graph.d_label6666 : graph.d_label5555);//cudamemcpy
				hipDeviceSynchronize();
				gpuErrorcheck( hipPeekAtLastError() );	
				// //hipStreamSynchronize(stream0);
				// hipStreamQuery(stream0);
				// //hipStreamSynchronize(stream1);
				// hipStreamQuery(stream1);
				gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished1, d_finished1, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished2, d_finished2, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished3, d_finished3, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished4, d_finished4, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished5, d_finished5, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished6, d_finished6, sizeof(bool), hipMemcpyDeviceToHost));
				gpuErrorcheck(hipMemcpy(&finished7, d_finished7, sizeof(bool), hipMemcpyDeviceToHost));
				hipDeviceSynchronize();
				hipStreamSynchronize(stream0);
				hipStreamSynchronize(stream1);
				hipStreamSynchronize(stream2);
				hipStreamSynchronize(stream3);
				hipStreamSynchronize(stream4);
				hipStreamSynchronize(stream5);
				hipStreamSynchronize(stream6);
				hipStreamSynchronize(stream7);
					//printf("^^^^^^^^^^^^^^^^^^^^\n");
					//cout<<"finished/finished1="<<finished<<""<<finished1<<endl;
			}while(!(finished)||!(finished1)||!(finished2)||!(finished3)||!(finished4)||!(finished5)||!(finished6)||!(finished7)); // finished=ture，finished=false才会循环 ,  ||finished1
			hipDeviceSynchronize();
			 hipStreamSynchronize(stream0);
			// hipStreamSynchronize(stream1);
			// hipEventRecord(stop, 0);
			// hipEventSynchronize(stop);
			// hipEventElapsedTime(&elapsedTime, start, stop);
			cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;
		}
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);
	hipStreamSynchronize(stream3);
	hipStreamSynchronize(stream4);
	hipStreamSynchronize(stream5);
	hipStreamSynchronize(stream6);
	hipStreamSynchronize(stream7);
		subgen.generate(graph, subgraph);  //这里应该只生成一个子图	
	}
				//gpuErrorcheck(hipMemcpyAsync(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice,stream0));

	float runtime = timer.Finish();
	cout << "Processing finished in " << runtime/1000 << " (s).\n";

	float readtime1 = timer1.Finish();
	cout << "Total finished in " << readtime1/1000 << " (s).\n";
	
	gpuErrorcheck(hipMemcpyAsync(graph.value, graph.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream0));
	gpuErrorcheck(hipMemcpyAsync(graph.value1, graph.d_value1, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream1));
	gpuErrorcheck(hipMemcpyAsync(graph.value2, graph.d_value2, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream2));
	gpuErrorcheck(hipMemcpyAsync(graph.value3, graph.d_value3, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream3));
	gpuErrorcheck(hipMemcpyAsync(graph.value4, graph.d_value4, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream4));
	gpuErrorcheck(hipMemcpyAsync(graph.value5, graph.d_value5, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream5));
	gpuErrorcheck(hipMemcpyAsync(graph.value6, graph.d_value6, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream6));
	gpuErrorcheck(hipMemcpyAsync(graph.value7, graph.d_value7, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream7));
	hipDeviceSynchronize();
	//gpuErrorcheck(hipMemcpyAsync(graph.value1, graph.d_value1, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost,stream1));
	printf("***************\n");
	// utilities::PrintResults(graph.value, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value1, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value2, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value3, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value4, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value5, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value6, min(30, graph.num_nodes));
	// utilities::PrintResults(graph.value7, min(30, graph.num_nodes));
	
	hipDeviceSynchronize();

	// for(int i=0; i<20; i++)
	// 	cout << graph.value[i] << endl;
			
	 if(arguments.hasOutput)
		utilities::SaveResults(arguments.output, graph.value, graph.num_nodes);
	// 	//utilities::SaveResults(arguments.output, graph.value1, graph.num_nodes);
	// }

}

