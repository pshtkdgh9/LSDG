#include "hip/hip_runtime.h"
#include "../shared/globals.hpp"
#include "../shared/timer.hpp"
#include "../shared/argument_parsing.cuh"
#include "../shared/graph.cuh"
#include "../shared/subgraph.cuh"
#include "../shared/partitioner.cuh"
#include "../shared/subgraph_generator.cuh"
#include "../shared/gpu_error_check.cuh"
#include "../shared/gpu_kernels.cuh"
#include "../shared/egraph_utilities.hpp"
#include "../shared/test.cuh"
#include "../shared/test.cu"
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <iostream>

int main()
{
	hipFree(0);

	unsigned int sourceNode = 0;
	bool hasOutput;
	string output;

	std::string input1 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot1.el";
    std::string input2 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot2.el";
    std::string input3 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot3.el";
    std::string input4 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot4.el";

    
    std::string input_log1 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot1_changes.log";
    std::string input_log2 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot2_changes.log";
    std::string input_log3 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot3_changes.log";
    std::string input_log4 = "/home/netdb/jihyeon/make_snapshot/sk-2005_snapshot4_changes.log";



	Timer timer;
	timer.Start();
	
	Graph<OutEdgeWeighted> graph1(input1, false);
	// graph1.ReadGraph();
    Graph<OutEdgeWeighted> graph2(input2, false);
	// graph2.ReadGraph();
    Graph<OutEdgeWeighted> graph3(input3, false);
	// graph3.ReadGraph();
    Graph<OutEdgeWeighted> graph4(input4, false);
	// graph4.ReadGraph();
	
	// float readtime = timer.Finish();
	// cout << "Graph Reading finished in " << readtime/1000 << " (s).\n";
	
	// for(unsigned int i=0; i<100; i++)
	// 	cout << graph1.edgeList[i].end << " " << graph1.edgeList[i].w8;
	
    //************* 연산 ************************
    
    
    
    //************* 연산 ************************
	std::vector<Graph<OutEdgeWeighted>> graphs = {graph1, graph2, graph3, graph4};
	float ProcessingResult;
	for(int i=0; i<4; i++)
	{	
		Graph<OutEdgeWeighted> graph = graphs[i];
		graph.ReadGraph();
		for(unsigned int i=0; i<graph.num_nodes; i++)
		{
			graph.value[i] = 0;
			graph.label1[i] = true;
			graph.label2[i] = false;
		}
		graph.value[sourceNode] = DIST_INFINITY;
		//graph.label[sourceNode] = true;


		gpuErrorcheck(hipMemcpy(graph.d_outDegree, graph.outDegree, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
		gpuErrorcheck(hipMemcpy(graph.d_value, graph.value, graph.num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
		gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
		gpuErrorcheck(hipMemcpy(graph.d_label2, graph.label2, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));
		
		Subgraph<OutEdgeWeighted> subgraph(graph.num_nodes, graph.num_edges);
		
		SubgraphGenerator<OutEdgeWeighted> subgen(graph);
		
		subgen.generate(graph, subgraph);
		
		for(unsigned int i=0; i<graph.num_nodes; i++)
		{
			graph.label1[i] = false;
		}
		graph.label1[sourceNode] = true;
		gpuErrorcheck(hipMemcpy(graph.d_label1, graph.label1, graph.num_nodes * sizeof(bool), hipMemcpyHostToDevice));	
		

		Partitioner<OutEdgeWeighted> partitioner;
		
		timer.Start();
		
		uint gItr = 0;
		
		bool finished;
		bool *d_finished;
		gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
			
		while (subgraph.numActiveNodes>0)
		{
			gItr++;
			
			partitioner.partition(subgraph, subgraph.numActiveNodes);
			// a super iteration
			for(int i=0; i<partitioner.numPartitions; i++)
			{
				hipDeviceSynchronize();
				gpuErrorcheck(hipMemcpy(subgraph.d_activeEdgeList, subgraph.activeEdgeList + partitioner.fromEdge[i], (partitioner.partitionEdgeSize[i]) * sizeof(OutEdgeWeighted), hipMemcpyHostToDevice));
				hipDeviceSynchronize();

				//moveUpLabels<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(subgraph.d_activeNodes, graph.d_label, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
				mixLabels<<<partitioner.partitionNodeSize[i]/512 + 1 , 512>>>(subgraph.d_activeNodes, graph.d_label1, graph.d_label2, partitioner.partitionNodeSize[i], partitioner.fromNode[i]);
				
				uint itr = 0;
				do
				{
					cout << "\t\tIteration " << ++itr << endl;
					finished = true;
					gpuErrorcheck(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));

					sswp_async<<< partitioner.partitionNodeSize[i]/512 + 1 , 512 >>>(partitioner.partitionNodeSize[i],
														partitioner.fromNode[i],
														partitioner.fromEdge[i],
														subgraph.d_activeNodes,
														subgraph.d_activeNodesPointer,
														subgraph.d_activeEdgeList,
														graph.d_outDegree,
														graph.d_value, 
														d_finished,
														(itr%2==1) ? graph.d_label1 : graph.d_label2,
														(itr%2==1) ? graph.d_label2 : graph.d_label1);	

					hipDeviceSynchronize();
					gpuErrorcheck( hipPeekAtLastError() );	
					
					gpuErrorcheck(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
				}while(!(finished));
				
				cout << itr << ((itr>1) ? " Inner Iterations" : " Inner Iteration") << " in Global Iteration " << gItr << ", Partition " << i  << endl;			
			}
			
			subgen.generate(graph, subgraph);
				
		}	
		
		float runtime = timer.Finish();
		cout << "Processing finished in " << runtime/1000 << " (s).\n";
		ProcessingResult += runtime;
		gpuErrorcheck(hipMemcpy(graph.value, graph.d_value, graph.num_nodes*sizeof(uint), hipMemcpyDeviceToHost));
		
		utilities::PrintResults(graph.value, min(30, graph.num_nodes));
				
		if(hasOutput)
			utilities::SaveResults(output, graph.value, graph.num_nodes);

		hipFree(0);
		}
		cout << "All Processing finished in " << ProcessingResult/1000 << " (s).\n";
}
